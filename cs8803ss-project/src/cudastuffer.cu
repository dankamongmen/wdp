#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <sys/mman.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "cuda8803ss.h"

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!

static int
init_cuda(int devno){
	int attr,cerr;
	hipCtx_t ctx;
	hipDevice_t c;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipDeviceGet(&c,devno)) != hipSuccess){
		fprintf(stderr,"Couldn't get device reference, exiting.\n");
		return cerr;
	}
	if((cerr = hipCtxCreate(&ctx,0,c)) != hipSuccess){
		fprintf(stderr,"Couldn't create context, exiting.\n");
		return cerr;
	}
	return hipSuccess;
}

// FIXME: we really ought take a bus specification rather than a device number,
// since the latter are unsafe across hardware removal/additions.
static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno\n",a0);
}

static int
get_devno(const char *argv0,const char *arg,unsigned long *zul){
	char *eptr;

	if(((*zul = strtoul(arg,&eptr,0)) == ULONG_MAX && errno == ERANGE)
			|| eptr == arg || *eptr){
		fprintf(stderr,"Invalid device number: %s\n",arg);
		usage(argv0);
		return -1;
	}
	return 0;
}

int main(int argc,char **argv){
	unsigned long zul;
	uintmax_t total,s;
	hipDeviceptr_t ptr;
	int cerr;

	if(argc != 2){
		usage(*argv);
		exit(EXIT_FAILURE);
	}
	if(get_devno(argv[0],argv[1],&zul)){
		exit(EXIT_FAILURE);
	}
	if((cerr = init_cuda(zul)) != hipSuccess){
		fprintf(stderr,"Error initializing CUDA device %d (%d, %s?)\n",
				zul,cerr,hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	if((s = cuda_alloc_max(stdout,1ul << ADDRESS_BITS,&ptr,sizeof(unsigned))) == 0){
		fprintf(stderr,"Error allocating max on device %d (%s?)\n",
			zul,hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
	total = s;
	while( (s = cuda_alloc_max(stdout,1ul << ADDRESS_BITS,&ptr,sizeof(unsigned))) ){
		total += s;
	}
	printf(" Got a total of %jub (0x%jx)\n",total,total);
	exit(EXIT_SUCCESS);
}
