#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,unsigned *mem){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	hipDevice_t c;
	void *str;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		free(str);
		return cerr;
	}
	if((cerr = hipDeviceTotalMem(mem,c)) != hipSuccess){
		return cerr;
	}
	printf("%d.%d %s %s %uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		*mem / (1024 * 1024),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "" :
		"(unknown compute mode)");
	free(str);
	return hipSuccess;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(unsigned *mem){
	int attr,count,z;
	int cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		/*if(cerr == hipErrorNoDevice){
			return 0;
		}*/
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(&count)) != hipSuccess){
		return cerr;
	}
	if(count == 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	printf("CUDA device count: %d\n",count);
	for(z = 0 ; z < count ; ++z){
		printf(" %03d ",z);
		if( (cerr = id_cuda(z,mem)) ){
			return cerr;
		}
	}
	return hipSuccess;
}

__global__ void memkernel(unsigned long *sum,unsigned b){
	unsigned bp;

	*sum = 0;
	for(bp = 0 ; bp < b ; ++bp){
		sum[0] += sum[bp];
	}
}

int main(void){
	unsigned long sum;
	unsigned mem;
	void *ptr;

	if(init_cuda(&mem)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
#define MASK 0x00fffffflu
	if(hipMalloc(&ptr,mem & MASK)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	memkernel<<<1,1>>>((typeof(&sum))ptr,(mem & MASK) / sizeof(sum));
	if(hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error running kernel (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	hipMemcpy(&sum,ptr,sizeof(sum),hipMemcpyDeviceToHost);
	printf(" sum: %u 0x%x\n",sum,sum);
	if(hipFree(ptr) || hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error dumping CUDA memory (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
