#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,unsigned *mem){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	hipDevice_t c;
	void *str;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		free(str);
		return cerr;
	}
	if((cerr = hipDeviceTotalMem(mem,c)) != hipSuccess){
		return cerr;
	}
	printf("%d.%d %s %s %uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		*mem / (1024 * 1024),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "" :
		"(unknown compute mode)");
	free(str);
	return hipSuccess;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(unsigned *mem){
	int attr,count,z;
	int cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(&count)) != hipSuccess){
		return cerr;
	}
	if(count == 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	printf("CUDA device count: %d\n",count);
	for(z = 0 ; z < count ; ++z){
		printf(" %03d ",z);
		if( (cerr = id_cuda(z,mem)) ){
			return cerr;
		}
	}
	return hipSuccess;
}

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!

__global__ void memkernel(unsigned long *sum,unsigned b){
	unsigned bp;

	*sum = 0;
	for(bp = 0 ; bp < b ; ++bp){
		sum[0] += *(unsigned long *)
			((unsigned long)(sum + bp) % (1lu << ADDRESS_BITS));
	}
}

int main(void){
	struct timeval time0,time1,timer;
	unsigned long sum;
	unsigned mem;
	void *ptr;

	if(init_cuda(&mem)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
#define CHUNK (mem >> 2u)
	printf(" Want %ub (0x%x) of %ub (0x%x)\n",mem - CHUNK,mem - CHUNK,mem,mem);
	if(hipMalloc(&ptr,mem - CHUNK)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	gettimeofday(&time0,NULL);
	memkernel<<<1,1>>>((typeof(&sum))ptr,(mem - CHUNK) / sizeof(sum));
	if(hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error running kernel (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	gettimeofday(&time1,NULL);
	timersub(&time1,&time0,&timer);
	hipMemcpy(&sum,ptr,sizeof(sum),hipMemcpyDeviceToHost);
	printf(" sum: %u 0x%x\n",sum,sum);
	printf(" elapsed time: %luus\n",timer.tv_sec * 1000000 + timer.tv_usec);
	if(hipFree(ptr) || hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error dumping CUDA memory (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
