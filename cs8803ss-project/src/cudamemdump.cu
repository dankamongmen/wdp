#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,unsigned *mem){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	hipDevice_t c;
	void *str;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		free(str);
		return cerr;
	}
	if((cerr = hipDeviceTotalMem(mem,c)) != hipSuccess){
		return cerr;
	}
	printf("%d.%d %s %s %uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		*mem / (1024 * 1024),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "" :
		"(unknown compute mode)");
	free(str);
	return hipSuccess;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(unsigned *mem){
	int attr,count,z;
	int cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(&count)) != hipSuccess){
		return cerr;
	}
	if(count == 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	printf("CUDA device count: %d\n",count);
	for(z = 0 ; z < count ; ++z){
		printf(" %03d ",z);
		if( (cerr = id_cuda(z,mem)) ){
			return cerr;
		}
	}
	return hipSuccess;
}

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!

#define BLOCK_SIZE 64 // FIXME bigger would likely be better

__global__ void memkernel(uintmax_t *sum,unsigned b){
	__shared__ typeof(*sum) psum;
	unsigned bp;

	psum = 0;
	for(bp = 0 ; bp < b ; bp += BLOCK_SIZE){
		psum += *(uintmax_t *)
			((uintmax_t)(sum + bp + threadIdx.x) % (1lu << ADDRESS_BITS));
	}
	sum[threadIdx.x] = psum;
}

int main(void){
	uintmax_t sums[BLOCK_SIZE],sum = 0;
	struct timeval time0,time1,timer;
	dim3 dblock(BLOCK_SIZE,1,1);
	unsigned mem;
	void *ptr;

	if(init_cuda(&mem)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
#define CHUNK (mem >> 2u)
	printf(" Want %ub (0x%x) of %ub (0x%x)\n",mem - CHUNK,mem - CHUNK,mem,mem);
	if(hipMalloc(&ptr,mem - CHUNK)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	gettimeofday(&time0,NULL);
	memkernel<<<1,dblock>>>((typeof(&sum))ptr,(mem - CHUNK) / sizeof(*sums));
	if(hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error running kernel (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	gettimeofday(&time1,NULL);
	timersub(&time1,&time0,&timer);
	hipMemcpy(sums,ptr,sizeof(sums),hipMemcpyDeviceToHost);
	for(int i = 0 ; i < BLOCK_SIZE ; ++i){
		sum += sums[i];
	}
	printf(" sum: %ju 0x%jx\n",sum,sum);
	printf(" elapsed time: %luus (%.3f Mb/s)\n",
			timer.tv_sec * 1000000 + timer.tv_usec,
			(float)(mem - CHUNK) / (timer.tv_sec * 1000000 + timer.tv_usec));
	if(hipFree(ptr) || hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error dumping CUDA memory (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
