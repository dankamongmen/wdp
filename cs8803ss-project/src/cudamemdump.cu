#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,unsigned *mem){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	hipDevice_t c;
	void *str;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		free(str);
		return cerr;
	}
	if((cerr = hipDeviceTotalMem(mem,c)) != hipSuccess){
		return cerr;
	}
	printf("%d.%d %s %s %uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		*mem / (1024 * 1024),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "" :
		"(unknown compute mode)");
	free(str);
	return hipSuccess;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(int *count){
	int attr,cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(count)) != hipSuccess){
		return cerr;
	}
	if(*count <= 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	printf("CUDA device count: %d\n",*count);
	return hipSuccess;
}

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!
#define CHUNK (mem >> 2u) // FIXME kill
#define BLOCK_SIZE 64 // FIXME bigger would likely be better

__global__ void memkernel(unsigned *sum,unsigned b){
	__shared__ typeof(*sum) psum;
	typeof(sum) ptr;
	unsigned bp;

	psum = 0;
	for(ptr = (unsigned *)0x10000u ; ptr < sum ; ptr += BLOCK_SIZE){
		psum += ptr[threadIdx.x];
	}
	for(bp = 0 ; bp < b ; bp += BLOCK_SIZE){
		psum += *(typeof(sum))
			((uintmax_t)(sum + bp + threadIdx.x) % (1lu << ADDRESS_BITS));
	}
	sum[threadIdx.x] = psum;
}

static int
dump_cuda(unsigned mem){
	unsigned sums[BLOCK_SIZE],sum = 0;
	struct timeval time0,time1,timer;
	dim3 dblock(BLOCK_SIZE,1,1);
	void *ptr;

	printf(" Want %ub (0x%x) of %ub (0x%x)\n",mem - CHUNK,mem - CHUNK,mem,mem);
	if(hipMalloc(&ptr,mem - CHUNK)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	printf(" Allocated %u MB at %p\n",(mem - CHUNK) / (1024 * 1024),ptr);
	gettimeofday(&time0,NULL);
	memkernel<<<1,dblock>>>((typeof(&sum))ptr,(mem - CHUNK) / sizeof(*sums));
	if(hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error running kernel (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	hipMemcpy(sums,ptr,sizeof(sums),hipMemcpyDeviceToHost);
	for(int i = 0 ; i < BLOCK_SIZE ; ++i){
		sum += sums[i];
	}
	gettimeofday(&time1,NULL);
	timersub(&time1,&time0,&timer);
	printf(" sum: %u\n",sum);
	printf(" elapsed time: %luus (%.3f MB/s)\n",
			timer.tv_sec * 1000000 + timer.tv_usec,
			(float)(mem - CHUNK) / (timer.tv_sec * 1000000 + timer.tv_usec));
	if(hipFree(ptr) || hipDeviceSynchronize()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error dumping CUDA memory (%s?)\n",
				hipGetErrorString(err));
		return -1;
	}
	return 0;
}

int main(void){
	int z,count;

	if(init_cuda(&count)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	for(z = 0 ; z < count ; ++z){
		unsigned mem;

		printf(" %03d ",z);
		if(id_cuda(z,&mem)){
			hipError_t err;

			err = hipGetLastError();
			fprintf(stderr,"\nError probing CUDA device %d (%s?)\n",
					z,hipGetErrorString(err));
			return EXIT_FAILURE;
		}
		if(dump_cuda(mem)){
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
