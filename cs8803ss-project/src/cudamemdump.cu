#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	unsigned mem;
	hipDevice_t c;
	void *str;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		free(str);
		return cerr;
	}
	if((cerr = hipDeviceTotalMem(&mem,c)) != hipSuccess){
		return cerr;
	}
	printf("%d.%d %s %s %uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		mem / (1024 * 1024),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "" :
		"(unknown compute mode)");
	free(str);
	return hipSuccess;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(void){
	int attr,count,z;
	int cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		/*if(cerr == hipErrorNoDevice){
			return 0;
		}*/
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(&count)) != hipSuccess){
		return cerr;
	}
	if(count == 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	printf("CUDA device count: %d\n",count);
	for(z = 0 ; z < count ; ++z){
		printf(" %03d ",z);
		if( (cerr = id_cuda(z)) ){
			return cerr;
		}
	}
	return hipSuccess;
}

__global__ void memkernel(unsigned long *sum,unsigned long *words){
	const unsigned long *mem;
	int i;

	mem = sum;
	for(i = 0 ; i < 0x20000 ; ++i){
		*sum += *mem++;
		++*words;
	}
}

int main(void){
	unsigned long sum = 0,words = 0;
	void *ptr;

	if(init_cuda()){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	if(hipMalloc(&ptr,sizeof(sum) * 2)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	hipMemset(ptr,0,sizeof(sum) * 2);
	memkernel<<<1,1>>>((typeof(&sum))ptr,(typeof(&sum))ptr + 1);
	hipMemcpy(&sum,ptr,sizeof(sum),hipMemcpyDeviceToHost);
	hipMemcpy(&words,(typeof(&sum))ptr + 1,sizeof(sum),hipMemcpyDeviceToHost);
	printf("sum: %u 0x%x\nwords: %u 0x%x (%u 0x%x bytes)\n",
			sum,sum,words,words,
			words * sizeof(sum),
			words * sizeof(sum));
	if(hipFree(ptr) || words == 0){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error dumping CUDA memory (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
