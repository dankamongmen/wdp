#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#define BLOCK_SIZE 64

static int
dumpresults(const uint32_t *res,unsigned count){
	unsigned z,y;

	for(z = 0 ; z < count ; z += 8){
		for(y = 0 ; y < 8 ; ++y){
			if(printf("%9u ",res[z + y]) < 0){
				return -1;
			}
		}
		if(printf("\n") < 0){
			return -1;
		}
	}
	return 0;
}

__global__ void
memkernel(unsigned *aptr,unsigned *bptr,unsigned *results){
	__shared__ typeof(*results) psum[BLOCK_SIZE];
	unsigned z;

	psum[threadIdx.x] = results[threadIdx.x];
	while(aptr + threadIdx.x < bptr){
		//printf("%p %u\n",(unsigned *)(aptr + unit * threadIdx.x),*(unsigned *)(aptr + unit * threadIdx.x));
		if(aptr[threadIdx.x]){
			++psum[threadIdx.x];
		}
		aptr += BLOCK_SIZE;
	}
	results[threadIdx.x] = psum[threadIdx.x];
	__syncthreads();
}

__global__ void
cudasanity(uint32_t *res,unsigned byte){
	__shared__ uint32_t psum[BLOCK_SIZE];

	psum[threadIdx.x] = res[threadIdx.x];
	psum[threadIdx.x] = byte;
	res[threadIdx.x] = psum[threadIdx.x];
}

int main(void){
	unsigned hr[BLOCK_SIZE],*ptr;
	dim3 dblock(BLOCK_SIZE,1,1);
	dim3 dgrid(1,1);

	memset(hr,0,sizeof(hr));
	if(hipMalloc(&ptr,sizeof(hr)) || hipMemset(ptr,0x00,sizeof(hr))){
		return EXIT_FAILURE;
	}
//	cudasanity<<<dgrid,dblock>>>(ptr,0xf0);
	memkernel<<<dgrid,dblock>>>(ptr,(unsigned *)((char *)ptr + sizeof(hr)),ptr);
	if(hipMemcpy(hr,ptr,sizeof(hr),hipMemcpyDeviceToHost)){
		return EXIT_FAILURE;
	}
	if(hipFree(ptr)){
		return EXIT_FAILURE;
	}
	if(dumpresults(hr,sizeof(hr) / sizeof(*hr))){
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
