#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda8803ss.h"

static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno\n",a0);
}

__global__ void
touchbytes(hipDeviceptr_t ptr,uint32_t off,hipDeviceptr_t res){
	uint8_t b;

	b = *(unsigned char *)((uintptr_t)ptr + off + blockIdx.x);
	if(b == 0xff){
		*(uint32_t *)((uintptr_t)res + blockIdx.x) = 1;
	}
}

#define BYTES_PER_KERNEL 4

static int
basic_params(hipDeviceptr_t p,size_t s){
	hipDeviceptr_t p2;
	hipError_t cerr;

	if( (cerr = hipMalloc(&p2,s)) || (cerr = hipMemsetD8(p2,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		return -1;
	}
	printf("Got secondary %zub allocation at %p\n",s,p2);
	if( (cerr = hipFree(p2)) ){
		fprintf(stderr,"Couldn't free %zu base (%d)\n",s,cerr);
		return -1;
	}
	// FIXME not very rigorous, not at all...[frown]
	printf("Minimum cuMalloc() alignment might be: %u\n",p2 - p);
	return 0;
}

int main(int argc,char **argv){
	hipDeviceptr_t ptr,res;
	unsigned long zul;
	hipCtx_t ctx;
	hipError_t cerr;
	size_t s;

	if(argc != 2 || getzul(argv[1],&zul)){
		usage(*argv);
		exit(EXIT_FAILURE);
	}
	if(init_cuda_ctx(zul,&ctx)){
		exit(EXIT_FAILURE);
	}
	s = sizeof(ptr);
	if( (cerr = hipMalloc(&ptr,s)) || (cerr = hipMemsetD8(ptr,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		exit(EXIT_FAILURE);
	}
	printf("Got base %zub allocation at %p\n",s,ptr);
	if(basic_params(ptr,s)){
		exit(EXIT_FAILURE);
	}
	if( (cerr = hipMalloc(&res,BYTES_PER_KERNEL * sizeof(uint32_t))) ||
			(cerr = hipMemsetD32(res,0,BYTES_PER_KERNEL)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		exit(EXIT_FAILURE);
	}
	printf("Got result %zub allocation at %p\n",BYTES_PER_KERNEL * sizeof(uint32_t),res);
	exit(EXIT_SUCCESS);
}
