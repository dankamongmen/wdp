#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda8803ss.h"

static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno\n",a0);
}

static int
basic_params(hipDeviceptr_t p,size_t s){
	hipDeviceptr_t p2;
	hipError_t cerr;

	if( (cerr = hipMalloc(&p2,s)) || (cerr = hipMemsetD8(p2,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		return -1;
	}
	printf("Got secondary %zub allocation at %p\n",s,p2);
	if( (cerr = hipFree(p2)) ){
		fprintf(stderr,"Couldn't free %zu base (%d)\n",s,cerr);
		return -1;
	}
	// FIXME not very rigorous, not at all...[frown]
	printf("Minimum cuMalloc() alignment might be: %u\n",p2 - p);
	return 0;
}

#define BYTES_PER_KERNEL 4

__global__ void
touchbytes(hipDeviceptr_t ptr,uint32_t off,hipDeviceptr_t res){
	uint8_t b;

	b = *(unsigned char *)((uintptr_t)ptr + off + blockIdx.x);
	if(b == 0xff){
		*(uint32_t *)((uintptr_t)res + blockIdx.x * BYTES_PER_KERNEL) = 1;
	}
}

int main(int argc,char **argv){
	hipDeviceptr_t ptr,res;
	unsigned long zul;
	hipCtx_t ctx;
	hipError_t cerr;
	size_t s,z;

	if(argc != 2 || getzul(argv[1],&zul)){
		usage(*argv);
		exit(EXIT_FAILURE);
	}
	if(init_cuda_ctx(zul,&ctx)){
		exit(EXIT_FAILURE);
	}
	s = sizeof(ptr);
	if( (cerr = hipMalloc(&ptr,s)) || (cerr = hipMemsetD8(ptr,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		exit(EXIT_FAILURE);
	}
	printf("Got base %zub allocation at %p\n",s,ptr);
	if(basic_params(ptr,s)){
		exit(EXIT_FAILURE);
	}
	if( (cerr = hipMalloc(&res,BYTES_PER_KERNEL * sizeof(uint32_t))) ||
			(cerr = hipMemsetD32(res,0,BYTES_PER_KERNEL)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		exit(EXIT_FAILURE);
	}
	if(res <= ptr){ // FIXME...see loop detect below
		fprintf(stderr,"Unexpected pointer arrangement (%p >= %p)\n",ptr,res);
		exit(EXIT_FAILURE);
	}
	printf("Got result %zub allocation at %p\n",BYTES_PER_KERNEL * sizeof(uint32_t),res);
	z = 0;
	while((cerr = hipCtxSynchronize()) == hipSuccess){
		dim3 dg(1,1,1),db(BYTES_PER_KERNEL,1,1);

		printf("running on %zu\n",z);
		touchbytes<<<dg,db>>>(ptr,z,res);
		// FIXME check res
		if(((z += BYTES_PER_KERNEL) + ptr) > res){
			printf("Hit result array at %p; breaking loop\n",res);
			break;
		}
	}
	printf("Exited loop (ret: %d) at %zu\n",cerr,z);
	exit(EXIT_SUCCESS);
}
