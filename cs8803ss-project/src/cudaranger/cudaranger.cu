#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/mman.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!
#define CONSTWIN ((unsigned *)0x10000u)
#define BLOCK_SIZE 512

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,hipCtx_t *ctx){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	unsigned mem,tmem;
	void *str = NULL;
	hipDevice_t c;

	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeWarpSize,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return cerr;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return cerr;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		goto err;
	}
	if((cerr = hipCtxCreate(ctx,hipDeviceScheduleBlockingSync|hipDeviceScheduleYield,c)) != hipSuccess){
		goto err;
	}
	if((cerr = hipMemGetInfo(&mem,&tmem)) != hipSuccess){
		cuCtxDetach(*ctx);
		goto err;
	}
	if(printf(" %d.%d %s %s %u/%uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		mem / (1024 * 1024) + !!(mem / (1024 * 1024)),
		tmem / (1024 * 1024) + !!(tmem / (1024 * 1024)),
		dprop.computeMode == hipComputeModeExclusive ? "(exclusive)" :
		dprop.computeMode == hipComputeModeProhibited ? "(prohibited)" :
		dprop.computeMode == hipComputeModeDefault ? "(shared)" :
		"(unknown compute mode)") < 0){
		cuCtxDetach(*ctx);
		cerr = -1;
		goto err;
	}
	free(str);
	return hipSuccess;

err:	// cerr ought already be set!
	free(str);
	return cerr;
}

#define CUDAMAJMIN(v) v / 1000, v % 1000

static int
init_cuda(unsigned *count){
	int attr,cerr,c;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	printf("Compiled against CUDA version %d.%d. Linked against CUDA version %d.%d.\n",
			CUDAMAJMIN(CUDA_VERSION),CUDAMAJMIN(attr));
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipGetDeviceCount(&c)) != hipSuccess){
		return cerr;
	}
	if(c <= 0){
		fprintf(stderr,"No CUDA devices found, exiting.\n");
		return -1;
	}
	*count = c;
	printf("CUDA device count: %d\n",*count);
	return hipSuccess;
}

__global__ void
memkernel(uintptr_t aptr,const uintptr_t bptr,const unsigned unit){
	__shared__ unsigned psum[BLOCK_SIZE];

	psum[threadIdx.x] = 0;
	while(aptr + threadIdx.x * unit < bptr){
		psum[threadIdx.x] += *(unsigned *)(aptr + unit * threadIdx.x);
		aptr += BLOCK_SIZE * unit;
	}
}

static int
dump_cuda(uintmax_t tmin,uintmax_t tmax,unsigned unit){
	struct timeval time0,time1,timer;
	dim3 dblock(BLOCK_SIZE,1,1);
	int punit = 'M',cerr;
	dim3 dgrid(1,1,1);
	uintmax_t usec,s;
	float bw;

	if(tmin >= tmax){
		return -1;
	}
	s = tmax - tmin;
	printf("   memkernel {%u x %u} x {%u x %u x %u} (0x%jx, 0x%jx (%jub), %u)\n",
		dgrid.x,dgrid.y,dblock.x,dblock.y,dblock.z,tmin,tmax,s,unit);
	gettimeofday(&time0,NULL);
	memkernel<<<dgrid,dblock>>>(tmin,tmax,unit);
	if( (cerr = hipCtxSynchronize()) ){
		fprintf(stderr,"   Error running kernel (%d?)\n",cerr);
		return -1;
	}
	gettimeofday(&time1,NULL);
	timersub(&time1,&time0,&timer);
	usec = (timer.tv_sec * 1000000 + timer.tv_usec);
	bw = (float)s / usec;
	if(bw > 1000.0f){
		bw /= 1000.0f;
		punit = 'G';
	}
	printf("   elapsed time: %ju.%jus (%.3f %cB/s)\n",
			usec / 1000000,usec % 1000000,bw,punit);
	return 0;
}

// FIXME: we really ought take a bus specification rather than a device number,
// since the latter are unsafe across hardware removal/additions.
static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno addrmin addrmax\n",a0);
}

int main(int argc,char **argv){
	unsigned long long min,max;
	unsigned unit = 4;		// Minimum alignment of references
	unsigned long zul;
	unsigned count;
	hipError_t cerr;
	hipCtx_t ctx;
	char *eptr;

	if(argc != 4){
		usage(*argv);
		return EXIT_FAILURE;
	}
	if(((zul = strtoul(argv[1],&eptr,0)) == ULONG_MAX && errno == ERANGE)
			|| eptr == argv[1] || *eptr){
		fprintf(stderr,"Invalid device number: %s\n",argv[1]);
		usage(*argv);
		return EXIT_FAILURE;
	}
	if(((min = strtoull(argv[2],&eptr,0)) == ULLONG_MAX && errno == ERANGE)
			|| eptr == argv[2] || *eptr){
		fprintf(stderr,"Invalid minimum address: %s\n",argv[2]);
		usage(*argv);
		return EXIT_FAILURE;
	}
	if(((max = strtoull(argv[3],&eptr,0)) == ULLONG_MAX && errno == ERANGE)
			|| eptr == argv[3] || *eptr){
		fprintf(stderr,"Invalid maximum address: %s\n",argv[3]);
		usage(*argv);
		return EXIT_FAILURE;
	}
	if(init_cuda(&count)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"Error initializing CUDA (%s?)\n",
				hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	if(zul >= count){
		fprintf(stderr,"devno too large (%lu >= %d)\n",zul,count);
		usage(*argv);
		return EXIT_FAILURE;
	}
	if(id_cuda(zul,&ctx)){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\nError probing CUDA device %lu (%s?)\n",
				zul,hipGetErrorString(err));
		return EXIT_FAILURE;
	}
	if(dump_cuda(min,max,unit)){
		cuCtxDetach(ctx);
		return EXIT_FAILURE;
	}
	if((cerr = cuCtxDetach(ctx)) != hipSuccess){
		fprintf(stderr,"\nError detaching context (%d?)\n",cerr);
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}
