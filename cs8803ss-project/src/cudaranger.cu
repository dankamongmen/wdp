#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <sys/mman.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "cuda8803ss.h"

#define ADDRESS_BITS 32u // FIXME 40 on compute capability 2.0!

static int
init_cuda(int devno){
	int attr,cerr;
	hipCtx_t ctx;
	hipDevice_t c;

	if((cerr = hipInit(0)) != hipSuccess){
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipDeviceGet(&c,devno)) != hipSuccess){
		fprintf(stderr,"Couldn't get device reference, exiting.\n");
		return cerr;
	}
	if((cerr = hipCtxCreate(&ctx,hipDeviceScheduleBlockingSync|hipDeviceScheduleYield,c)) != hipSuccess){
		fprintf(stderr,"Couldn't create context, exiting.\n");
		return cerr;
	}
	return hipSuccess;
}

static int
dumpresults(const uint32_t *res,unsigned count){
	unsigned z,y,nonzero;

	nonzero = 0;
	for(z = 0 ; z < count ; z += 8){
		for(y = 0 ; y < 8 ; ++y){
			if(printf("%9x ",res[z + y]) < 0){
				return -1;
			}
			if(res[z + y]){
				++nonzero;
			}
		}
		if(printf("\n") < 0){
			return -1;
		}
	}
	if(nonzero == 0){
		fprintf(stderr,"  All-zero results. Kernel probably didn't run.\n");
		return -1;
	}
	return 0;
}

// FIXME: we really ought take a bus specification rather than a device number,
// since the latter are unsafe across hardware removal/additions.
static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno addrmin addrmax\n",a0);
}

int main(int argc,char **argv){
	uint32_t hostres[GRID_SIZE * BLOCK_SIZE],*resarr;
	unsigned long long min,max;
	unsigned unit = 4;		// Minimum alignment of references
	unsigned long zul;
	hipDeviceptr_t ptr;
	cudadump_e res;
	char *eptr;
	int cerr;

	if(argc != 4){
		usage(*argv);
		return CUDARANGER_EXIT_ERROR;
	}
	if(((zul = strtoul(argv[1],&eptr,0)) == ULONG_MAX && errno == ERANGE)
			|| eptr == argv[1] || *eptr){
		fprintf(stderr,"Invalid device number: %s\n",argv[1]);
		printf("%lu %d\n",zul,*eptr);
		usage(*argv);
		return CUDARANGER_EXIT_ERROR;
	}
	if(((min = strtoull(argv[2],&eptr,0)) == ULLONG_MAX && errno == ERANGE)
			|| eptr == argv[2] || *eptr){
		fprintf(stderr,"Invalid minimum address: %s\n",argv[2]);
		usage(*argv);
		return CUDARANGER_EXIT_ERROR;
	}
	if(((max = strtoull(argv[3],&eptr,0)) == ULLONG_MAX && errno == ERANGE)
			|| eptr == argv[3] || *eptr){
		fprintf(stderr,"Invalid maximum address: %s\n",argv[3]);
		usage(*argv);
		return CUDARANGER_EXIT_ERROR;
	}
	if(max <= min){
		fprintf(stderr,"Invalid arguments: max (%ju) <= min (%ju)\n",
				max,min);
		usage(*argv);
		return CUDARANGER_EXIT_ERROR;
	}
	if((cerr = init_cuda(zul)) != hipSuccess){
		fprintf(stderr,"Error initializing CUDA device %d (%d, %s?)\n",
				zul,cerr,hipGetErrorString(hipGetLastError()));
		return CUDARANGER_EXIT_ERROR;
	}
	if(hipMalloc(&resarr,sizeof(hostres)) || hipMemset(resarr,0x00,sizeof(hostres))){
		fprintf(stderr,"Error allocating %zu on device %d (%s?)\n",
			sizeof(hostres),zul,hipGetErrorString(hipGetLastError()));
		return CUDARANGER_EXIT_ERROR;
	}
	if(cuda_alloc_max(NULL,1ul << ADDRESS_BITS,&ptr,sizeof(unsigned)) == 0){
		fprintf(stderr,"Error allocating max on device %d (%s?)\n",
			zul,hipGetErrorString(hipGetLastError()));
		return CUDARANGER_EXIT_ERROR;
	}
	if((res = dump_cuda(min,max,unit,resarr)) != CUDARANGER_EXIT_SUCCESS){
		return res;
	}
	if(hipDeviceSynchronize()){
		return res;
	}
	if(hipFree(ptr)){
		fprintf(stderr,"Warning: couldn't free memory\n");
	}
	if(hipMemcpy(hostres,resarr,sizeof(hostres),hipMemcpyDeviceToHost)){
		fprintf(stderr,"Error copying %zu from device %d (%s?)\n",
			sizeof(hostres),zul,hipGetErrorString(hipGetLastError()));
		return CUDARANGER_EXIT_ERROR;
	}
	if(hipFree(resarr)){
		fprintf(stderr,"Couldn't free %zu on device %d (%s?)\n",
			sizeof(hostres),zul,hipGetErrorString(hipGetLastError()));
		return CUDARANGER_EXIT_ERROR;
	}
	if(dumpresults(hostres,sizeof(hostres) / sizeof(*hostres))){
		return CUDARANGER_EXIT_ERROR;
	}
	return CUDARANGER_EXIT_SUCCESS;
}
