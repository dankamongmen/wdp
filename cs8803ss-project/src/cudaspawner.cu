#include <hip/hip_runtime.h>
#include <errno.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <pthread.h>
#include <sys/types.h>

static unsigned thrdone,threadsmaintain = 1;
static pthread_cond_t cond = PTHREAD_COND_INITIALIZER;
static pthread_mutex_t lock = PTHREAD_MUTEX_INITIALIZER;

static int
init_thread(hipCtx_t *pctx,hipDevice_t dev,size_t s){
	hipDeviceptr_t ptr;
	hipError_t cerr;

	if( (cerr = hipCtxCreate(pctx,0,dev)) ){
		fprintf(stderr," Error (%d) creating CUDA context\n",cerr);
		return -1;
	}
	if(s){
		if( (cerr = hipMalloc(&ptr,s)) ){
			fprintf(stderr," Error (%d) allocating %zub\n",cerr,s);
			return -1;
		}
	}
	return 0;
}

typedef struct ctx {
	size_t s;
	hipDevice_t dev;
	unsigned threadno;
} ctx;

static void *
thread(void *unsafectx){
	ctx x = *(ctx *)unsafectx;
	hipCtx_t cu;

	if(init_thread(&cu,x.dev,x.s)){
		goto err;
	}
	pthread_mutex_lock(&lock);
	printf("Got context at %p\n",cu);
	thrdone = 1;
	pthread_cond_broadcast(&cond);
	while(threadsmaintain){
		pthread_cond_wait(&cond,&lock);
	}
	pthread_mutex_unlock(&lock);
	return NULL;

err:
	pthread_mutex_lock(&lock);
	thrdone = 1;
	threadsmaintain = 0;
	pthread_cond_broadcast(&cond);
	pthread_mutex_unlock(&lock);
	return NULL;
}

// FIXME: we really ought take a bus specification rather than a device number,
// since the latter are unsafe across hardware removal/additions.
static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno perthreadbytes\n",a0);
}

static int
get_devno(const char *argv0,const char *arg,unsigned long *zul){
	char *eptr;

	if(((*zul = strtoul(arg,&eptr,0)) == ULONG_MAX && errno == ERANGE)
			|| eptr == arg || *eptr){
		fprintf(stderr,"Invalid device number: %s\n",arg);
		usage(argv0);
		return -1;
	}
	return 0;
}

static int
init_cuda(int devno,hipDevice_t *c){
	int attr,cerr;

	if((cerr = hipInit(0)) != hipSuccess){
		fprintf(stderr,"Error (%d) initializing CUDA\n",cerr);
		return cerr;
	}
	if((cerr = hipDriverGetVersion(&attr)) != hipSuccess){
		return cerr;
	}
	if(CUDA_VERSION > attr){
		fprintf(stderr,"Compiled against a newer version of CUDA than that installed, exiting.\n");
		return -1;
	}
	if((cerr = hipDeviceGet(c,devno)) != hipSuccess){
		fprintf(stderr,"Couldn't get device reference, exiting.\n");
		return cerr;
	}
	return hipSuccess;
}

int main(int argc,char **argv){
	unsigned long zul;
	unsigned total = 0;
	ctx marsh;

	if(argc != 3){
		usage(argv[0]);
		exit(EXIT_FAILURE);
	}
	if(get_devno(argv[0],argv[1],&zul)){
		usage(argv[0]);
		exit(EXIT_FAILURE);
	}
	if(get_devno(argv[0],argv[2],&marsh.s)){
		usage(argv[0]);
		exit(EXIT_FAILURE);
	}
	if(init_cuda(zul,&marsh.dev)){
		exit(EXIT_FAILURE);
	}
	while( (marsh.threadno = ++total) ){
		pthread_t tid;
		int err;

		if( (err = pthread_create(&tid,NULL,thread,&marsh)) ){
			fprintf(stderr,"Couldn't create thread (%s?)\n",
					strerror(err));
			exit(EXIT_SUCCESS);
		}
		pthread_mutex_lock(&lock);
		while(!thrdone && threadsmaintain){
			pthread_cond_wait(&cond,&lock);
		}
		thrdone = 0;
		if(!threadsmaintain){
			pthread_mutex_unlock(&lock);
			break;
		}
		pthread_mutex_unlock(&lock);
		printf("Created thread %d\n",total);
	}	
	exit(EXIT_SUCCESS);
}
